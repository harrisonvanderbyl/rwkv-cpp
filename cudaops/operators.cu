#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>

#define MIN_VALUE (-1e38)
__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const double *__restrict__ const _w, const double *__restrict__ const _u, const double *__restrict__ const _k, const double *__restrict__ const _v,
                               double *__restrict__ const _y, double *__restrict__ const _aa, double *__restrict__ const _bb, double *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    double u = _u[_c];
    double w = _w[_c];
    const double *__restrict__ const k = _k + _offset;
    const double *__restrict__ const v = _v + _offset;
    double *__restrict__ const y = _y + _offset;

    double aa = _aa[_state_offset];
    double bb = _bb[_state_offset];
    double pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;

        const double kk = exp(k[ii]);
        const double vv = v[ii];
        const double wr1 = aa +  exp(u+w+k[ii]) * vv;
        const double wr2 = bb +  exp(u+w+k[ii]);
        y[ii] = wr1 / wr2;
        aa = (aa + kk*vv) * exp(w);
        bb = (bb + kk) * exp(w);
        // const double kk = double(k[ii]);
        // const double vv = double(v[ii]);
        // double ww = u + kk;
        // double p = max(pp, ww);
        // double e1 = exp(pp - p);
        // double e2 = exp(ww - p);
        // y[ii] = ((e1 * aa + e2 * vv) / (e1 * bb + e2));
        // ww = w + pp;
        // p = max(ww, kk);
        // e1 = exp(ww - p);
        // e2 = exp(kk - p);
        // aa = e1 * aa + e2 * vv;
        // bb = e1 * bb + e2;
        // pp = p;

    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}

void cuda_wkv_forward(int B, int T, int C, double *w, double *u, double *k, double *v, double *y, double *aa, double *bb, double *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}




#define MM8_ONE_JSPLIT 16
#define MM8_ONE_TILE 1024

__device__ double atomicAddC(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void kernel_mm8_one(
    const int N, const int M,
    const double *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    double *__restrict__ const y,
    const double *__restrict__ const r
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        double y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j])
                
            );
        }
        atomicAddC(reinterpret_cast<double *>(&y[k]), *reinterpret_cast<double *>(&y_local));
    }
}



__global__ void kernel_mm8_three(
    const int N, const int M,
    const double *__restrict__ const x,
    const double *__restrict__ const x1,
    const double *__restrict__ const x2,

    const uint8_t *__restrict__ const w, const int w_stride,
    const uint8_t *__restrict__ const w1, const int w1_stride,
    const uint8_t *__restrict__ const w2, const int w2_stride,
    double *__restrict__ const y,
    double *__restrict__ const y1,
    double *__restrict__ const y2,
    const double *__restrict__ const r,
    const double *__restrict__ const r1,
    const double *__restrict__ const r2
    
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        double y_local = 0;
        double y1_local = 0;
        double y2_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j]));
            y1_local += x1[j] * (
                (w1[j * w1_stride + k] * r1[j]));
            y2_local += x2[j] * (
                (w2[j * w2_stride + k] * r2[j]));
           
        }
        atomicAddC(reinterpret_cast<double *>(&y[k]), *reinterpret_cast<double *>(&y_local));
        atomicAddC(reinterpret_cast<double *>(&y1[k]), *reinterpret_cast<double *>(&y1_local));
        atomicAddC(reinterpret_cast<double *>(&y2[k]), *reinterpret_cast<double *>(&y2_local));
    }
}
// generic T either float or fp16 or fp64

void cuda_mm8_three(int N, int M,
                    double *x,
                    double *x1,
                    double *x2,
                    uint8_t *w, int w_stride,
                    uint8_t *w1, int w1_stride,
                    uint8_t *w2, int w2_stride,
                    double *y,
                    double *y1,
                    double *y2,
                    double *r  ,
                    double *r1,
                    double *r2 
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_three<<<gridSize, blockSize>>>(
        N, M, x, x1, x2, w, w_stride, w1, w1_stride, w2, w2_stride, y, y1, y2, r, r1, r2);
}
void cuda_mm8_one(int N, int M,
                  double *x,
                  uint8_t *w, int w_stride,
                  double *y,
                    double *r   
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,y, r);
                }
