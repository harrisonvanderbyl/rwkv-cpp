#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#include <hip/hip_fp16.h>
#define MIN_VALUE (-1e38)
__global__ void kernel_wkv_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const float *__restrict__ const _k, const float *__restrict__ const _v,
                               float *__restrict__ const _y, float *__restrict__ const _aa, float *__restrict__ const _bb, float *__restrict__ const _pp) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _state_offset = _b * C + _c;

    float u = _u[_c];
    float w = _w[_c];
    const float *__restrict__ const k = _k + _offset;
    const float *__restrict__ const v = _v + _offset;
    float *__restrict__ const y = _y + _offset;

    float aa = _aa[_state_offset];
    float bb = _bb[_state_offset];
    float pp = _pp[_state_offset];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;

        const float kk = exp(k[ii]);
        const float vv = v[ii];
        const float wr1 = aa +  exp(u+w+k[ii]) * vv;
        const float wr2 = bb +  exp(u+w+k[ii]);
        y[ii] = wr1 / wr2;
        aa = (aa + kk*vv) * exp(w);
        bb = (bb + kk) * exp(w);
        // const float kk = float(k[ii]);
        // const float vv = float(v[ii]);
        // float ww = u + kk;
        // float p = max(pp, ww);
        // float e1 = exp(pp - p);
        // float e2 = exp(ww - p);
        // y[ii] = ((e1 * aa + e2 * vv) / (e1 * bb + e2));
        // ww = w + pp;
        // p = max(ww, kk);
        // e1 = exp(ww - p);
        // e2 = exp(kk - p);
        // aa = e1 * aa + e2 * vv;
        // bb = e1 * bb + e2;
        // pp = p;

    }
    _aa[_state_offset] = aa;
    _bb[_state_offset] = bb;
    _pp[_state_offset] = pp;
}

void cuda_wkv_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y, float *aa, float *bb, float *pp) {
    dim3 threadsPerBlock( min(C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_wkv_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, aa, bb, pp);
}




#define MM8_ONE_JSPLIT 16
#define MM8_ONE_TILE 1024



__global__ void kernel_mm8_one(
    const int N, const int M,
    const float *__restrict__ const x,
    const uint8_t *__restrict__ const w, const int w_stride,
    float *__restrict__ const y,
    const float *__restrict__ const r
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j])
                
            );
        }
        atomicAdd(reinterpret_cast<float *>(&y[k]), *reinterpret_cast<float *>(&y_local));
    }
}



__global__ void kernel_mm8_three(
    const int N, const int M,
    const float *__restrict__ const x,
    const float *__restrict__ const x1,
    const float *__restrict__ const x2,

    const uint8_t *__restrict__ const w, const int w_stride,
    const uint8_t *__restrict__ const w1, const int w1_stride,
    const uint8_t *__restrict__ const w2, const int w2_stride,
    float *__restrict__ const y,
    float *__restrict__ const y1,
    float *__restrict__ const y2,
    const float *__restrict__ const r,
    const float *__restrict__ const r1,
    const float *__restrict__ const r2
    
    ){

    const int k = blockIdx.y * blockDim.y + threadIdx.y;
    const int j0 = min(N, blockIdx.x * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));
    const int j1 = min(N, (blockIdx.x + 1) * ((N + MM8_ONE_JSPLIT - 1) / MM8_ONE_JSPLIT));

    if (k < M) {
        float y_local = 0;
        float y1_local = 0;
        float y2_local = 0;
        for (int j = j0; j < j1; ++j) {
            y_local += x[j] * (
                (w[j * w_stride + k] * r[j]));
            y1_local += x1[j] * (
                (w1[j * w1_stride + k] * r1[j]));
            y2_local += x2[j] * (
                (w2[j * w2_stride + k] * r2[j]));
           
        }
        atomicAdd(reinterpret_cast<float *>(&y[k]), *reinterpret_cast<float *>(&y_local));
        atomicAdd(reinterpret_cast<float *>(&y1[k]), *reinterpret_cast<float *>(&y1_local));
        atomicAdd(reinterpret_cast<float *>(&y2[k]), *reinterpret_cast<float *>(&y2_local));
    }
}
// generic T either float or fp16 or fp64

void cuda_mm8_three(int N, int M,
                    float *x,
                    float *x1,
                    float *x2,
                    uint8_t *w, int w_stride,
                    uint8_t *w1, int w1_stride,
                    uint8_t *w2, int w2_stride,
                    float *y,
                    float *y1,
                    float *y2,
                    float *r  ,
                    float *r1,
                    float *r2 
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_three<<<gridSize, blockSize>>>(
        N, M, x, x1, x2, w, w_stride, w1, w1_stride, w2, w2_stride, y, y1, y2, r, r1, r2);
}
void cuda_mm8_one(int N, int M,
                  float *x,
                  uint8_t *w, int w_stride,
                  float *y,
                    float *r   
                ) {
    dim3 blockSize(1, MM8_ONE_TILE);
    dim3 gridSize(MM8_ONE_JSPLIT, (M + blockSize.y - 1) / blockSize.y);
    kernel_mm8_one<<<gridSize, blockSize>>>(
        N, M, x, w, w_stride,y, r);
                }
